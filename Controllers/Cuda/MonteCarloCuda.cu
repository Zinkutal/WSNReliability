#include "hip/hip_runtime.h"
#include <cstdlib>
#include <ctime>
#include <vector>

#include "../../Models/Graph.h"

class MonteCarloCuda {
public:
    MonteCarloCuda(unsigned int accuracy, unsigned int oImgSizeX, unsigned int oImgSizeY, unsigned long kProvided,
                   unsigned long kConnected, unsigned long maxCoverage, float coverageFlag, Graph graphModel) :
            _accuracy(accuracy),
            _oImgSizeX(oImgSizeX),
            _oImgSizeY(oImgSizeY),
            _kProvided(kProvided),
            _kConnected(kConnected),
            _maxCoverage(maxCoverage),
            _coverageFlag(coverageFlag),
            _graphModel(graphModel){
        this->init();
    }

    void init() {
        srand(static_cast <unsigned> (time(0)));// Seeding random number
    }

    // Help function countSquareMatrix method
    void drawCircle(bool **matrix, int x0, int y0, int radius)
    {
        int x = radius;
        int y = 0;
        int xChange = 1 - (radius << 1);
        int yChange = 0;
        int radiusError = 0;

        while (x >= y)
        {
            for (int i = x0 - x; i <= x0 + x; i++)
            {
                if ((i >= 0) && (i < this->_oImgSizeX)) {
                    int yNew = y0 + y;
                    if ((yNew >= 0) && (yNew < this->_oImgSizeY)) matrix[i][yNew] = true;
                    yNew = y0 - y;
                    if ((yNew >= 0) && (yNew < this->_oImgSizeY)) matrix[i][yNew] = true;
                }
            }
            for (int i = x0 - y; i <= x0 + y; i++)
            {
                if ((i >= 0) && (i < this->_oImgSizeX)){
                    int yNew = y0 + x;
                    if ((yNew >= 0) && (yNew < this->_oImgSizeY)) matrix[i][yNew] = true;
                    yNew = y0 - x;
                    if ((yNew >= 0) && (yNew < this->_oImgSizeY)) matrix[i][yNew] = true;
                }
            }

            y++;
            radiusError += yChange;
            yChange += 2;
            if (((radiusError << 1) + xChange) > 0)
            {
                x--;
                radiusError += xChange;
                xChange += 2;
            }
        }
    }

    float countSquareMatrix(vector<float> visited){
        bool** matrix = new bool*[this->_oImgSizeX];
        for(int i = 0; i < this->_oImgSizeX; i++)
            matrix[i] = new bool[this->_oImgSizeY];

        int count_black = 0;

        // Initialize matrix
        for (unsigned int i=0; i < this->_oImgSizeX; i++) {
            for (unsigned int j=0; j < this->_oImgSizeY; j++) {
                matrix[i][j] = false;
            }
        }

        // Draw node circles
        for (int i=1; i < visited.size(); i++ ){
            if (visited.at(i) == 1) {
                int x = this->_graphModel.getNodes().at(i).getCoordinates().at(0);
                int y = this->_graphModel.getNodes().at(i).getCoordinates().at(1);
                int radius = this->_graphModel.getNodes().at(i).getCoverage() * this->_accuracy;
                this->drawCircle(matrix, x, y, radius);
            }
        }

        // Count covered area
        for (unsigned int i=0; i < this->_oImgSizeX; i++) {
            for (unsigned int j=0; j < this->_oImgSizeY; j++) {
                if (matrix[i][j]) {
                    count_black++;
                }
            }
        }

        // Debug Code
        /*for(int i = 0; i < this->_oImgSizeX; i++) {
            for (int j = 0; j < this->_oImgSizeY; j++)
                if (matrix[i][j]) { std::cout << "*"; } else std::cout << " ";
            std::cout << "\n";
        }
        std::cout << "\n --------------- \n";*/

        // Dealloc memory
        delete[] matrix;

        float square = count_black;
        int k = this->_maxCoverage;
        square /= k;

        return square;
    }

    // Return set of visited vertices in connected graph
    void recursiveVertexVisit(vector<float> nodeRel) {
        unsigned int v = 0;

        for (unsigned int i = 0; i < nodeRel.size(); i++) {
            if (nodeRel.at(i) == 1) {
                for (unsigned int neighborVertexId: this->_graphModel.getNodes().at(i).getRelations()) {
                    float visitedV = this->getVisitedNodes().at(neighborVertexId);
                    if ((nodeRel.at(neighborVertexId) > 0) && (visitedV != 1)) {
                        v = neighborVertexId;
                        break;
                    }
                }
            }
            if (v > 0) break;
        }

        if (v > 0) {
            vector<float> visited = this->getVisitedNodes();
            visited.at(v) = 1;
            this->setVisitedNodes(visited);
            this->recursiveVertexVisit(nodeRel);
        }
    }

    vector<float> updateGraphConnectivity(vector<float> nodeRel) {
        // Init & fill vector with non-visited vertices
        vector<float> visited;
        for (unsigned long i = 0; i < nodeRel.size(); i++) {
            visited.push_back(0);
        }
        visited.at(0) = 1; // Stock is always connected

        this->setVisitedNodes(visited);
        this->recursiveVertexVisit(nodeRel);

        visited = this->getVisitedNodes();

        for (unsigned long i = 1; i < visited.size(); i++) {
            if (visited.at(i) != 1) {
                nodeRel.at(i) = 0;
            }
        }

        return nodeRel;
    }

    float reliabilityParallelMethodMatrix(vector<float> nodeRel) {
        unsigned long kConn = this->getKConnected();
        vector<float> newRealization;

        unsigned long i, j;
        for (i = 0; i < this->_kProvided; i++) {
            newRealization = nodeRel;
            for (j = 1; j < nodeRel.size(); j++) {
                float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
                newRealization.at(j) = (nodeRel.at(j) >= r) ? 1 : 0;
            }
            newRealization = this->updateGraphConnectivity(newRealization);
            if (this->countSquareMatrix(newRealization) >= this->_coverageFlag) this->setKConnected(kConn++);
        }

        float result = this->getKConnected();
        result /= this->_kProvided;

        return result;
    }
    float reliabilityParallelExpectedMethodMatrix(vector<float> nodeRel) {
        float *kConnectedArr = new float[this->_kProvided];
        vector<float> newRealization;

        unsigned long i, j;
        for (i = 0; i < this->_kProvided; i++) {
            newRealization = nodeRel;
            for (j = 1; j < nodeRel.size(); j++) {
                float r = static_cast <float> (rand()) / static_cast <float> (RAND_MAX);
                newRealization.at(j) = (nodeRel.at(j) >= r) ? 1 : 0;
            }

            newRealization = this->updateGraphConnectivity(newRealization);
            kConnectedArr[i] = this->countSquareMatrix(newRealization);
        }

        float kConnected = 0, result = 0;
        for (unsigned long i = 0; i < this->_kProvided; i++) {
            kConnected += kConnectedArr[i];
        }
        result += kConnected;
        result /= this->_kProvided;

        return result;
    }
private:
    unsigned long getKConnected() {
        return this->_kConnected;
    }

    void setKConnected(unsigned long kConnected) {
        this->_kConnected = kConnected;
    }

    vector<float> getVisitedNodes() {
        return this->_visited;
    }

    void setVisitedNodes(vector<float> visited) {
        this->_visited = visited;
    }

    unsigned int _accuracy;
    unsigned int _oImgSizeX;
    unsigned int _oImgSizeY;
    unsigned long _kProvided;
    unsigned long _kConnected;
    unsigned long _maxCoverage;
    float _coverageFlag;
    vector<float> _visited;
    Graph _graphModel;
};